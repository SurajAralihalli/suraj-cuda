#include <iostream>
#include <hip/hip_runtime.h>

__global__ void busy() {
    int start = clock();
    while ((clock() - start) < 100'000'000);
    printf("I'm awake!\n");
}

int main()
{
    hipStream_t streams[5];
    for (int i = 0; i <5; i++) {
        hipStreamCreate(&streams[i]);
        busy<<<1, 1,0,streams[i]>>>();
    }

    hipDeviceSynchronize();
    
    for (int i = 0; i <5; i++) {
        hipStreamDestroy(streams[i]);
    }
}