#include <iostream>
#include <hip/hip_runtime.h>

__device__ float result = 0;

__global__ void reduceAtomicGlobal(const float* input, int N) {
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    if (id < N)
    atomicAdd(&result, input[id]);
}

int main() {
    int N = 4000;
    float* array = NULL;
    hipMallocManaged((float**)&array, sizeof(float) * N);
    
    for(int i=0;i<N;i++) {
        array[i] = i+0.5;
    }

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    reduceAtomicGlobal<<<gridSize,blockSize>>>(array, N);
    hipDeviceSynchronize();

    float hostResult;
    hipMemcpyFromSymbol(&hostResult, HIP_SYMBOL(result), sizeof(float));

    std::cout << "sum: " << hostResult << std::endl;

    hipFree(array);

}

